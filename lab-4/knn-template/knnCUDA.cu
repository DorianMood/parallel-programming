#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime_api.h>
#include <iostream>

#include <thrust/sort.h>

#include "knn.cu"

#define DEBUG 1

void check_error(hipError_t err, const char *msg);
void knnParallel(float *coords, float *newCoords, int *classes, int numClasses, int numSamples, int numNewSamples, int k);

void knnParallel(float *coords, float *newCoords, int *classes, int numClasses, int numSamples, int numNewSamples, int k)
{
    //*** Device-variables-declaration ***
    float *d_coords;
    float *d_newCoords;
    int *d_classes;
    float *d_distances;

    int totalSamples = numSamples + numNewSamples;

    //*** device-allocation ***
    check_error(hipMalloc(&d_coords, totalSamples * DIMENSION * sizeof(float)), "alloc d_coords_x");
    check_error(hipMalloc(&d_classes, totalSamples * sizeof(int)), "alloc d_classes");
    check_error(hipMalloc(&d_newCoords, numNewSamples * DIMENSION * sizeof(float)), "alloc d_coordsnew");
    check_error(hipMalloc(&d_distances, numNewSamples * numSamples * sizeof(float)), "alloc d_distances");

    //***copy-arrays-on-device***
    check_error(hipMemcpy(d_coords, coords, totalSamples * DIMENSION * sizeof(float), hipMemcpyHostToDevice), "copy d_coords");
    check_error(hipMemcpy(d_classes, classes, totalSamples * sizeof(int), hipMemcpyHostToDevice), "copy d_classes");
    check_error(hipMemcpy(d_newCoords, newCoords, numNewSamples * DIMENSION * sizeof(float), hipMemcpyHostToDevice), "copy d_coordsnew");

    const int PROBLEM_SIZE = numNewSamples;
    const int NUM_THREADS = 256;
    const int NUM_BLOCKS = (int)ceil(PROBLEM_SIZE / NUM_THREADS);

    // Calculate distances
    // For performance reasons we can compute just x^2 + y^2 (anyway we are looking for minimum)
    cuda_compute_distance<<<NUM_BLOCKS, NUM_THREADS>>>(d_coords, numSamples, d_newCoords, numNewSamples, d_distances);
    // Sort classes by distances
    //thrust::sort_by_key(d_distances, d_distances + 10, d_classes);

    // Calculate classes frequencies

    // Get class with the highest frequency

    // Write class to output



    hipDeviceSynchronize();
    // download device -> host
    check_error(hipMemcpy(coords, d_coords, DIMENSION * totalSamples * sizeof(float), hipMemcpyDeviceToHost), "download coords");
    check_error(hipMemcpy(classes, d_classes, totalSamples * sizeof(int), hipMemcpyDeviceToHost), "download classes");


}

void check_error(hipError_t err, const char *msg)
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "%s : error %d (%s)\n", msg, err, hipGetErrorString(err));
        exit(err);
    }
}
