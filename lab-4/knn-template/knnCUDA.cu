#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime_api.h>
#include <iostream>

#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

#include "knn.cu"

#define DEBUG 1

void check_error(hipError_t err, const char *msg);
void knnParallel(float *coords, float *newCoords, int *classes, int numClasses, int numSamples, int numNewSamples, int k);

void knnParallel(float *coords, float *newCoords, int *classes, int numClasses, int numSamples, int numNewSamples, int k)
{
    //*** Device-variables-declaration ***
    float *d_coords;
    float *d_newCoords;
    int *d_classes;
    float *d_distances;

    int totalSamples = numSamples + numNewSamples;

    //*** device-allocation ***
    check_error(hipMalloc(&d_coords, totalSamples * DIMENSION * sizeof(float)), "alloc d_coords_x");
    check_error(hipMalloc(&d_classes, totalSamples * sizeof(int)), "alloc d_classes");
    check_error(hipMalloc(&d_newCoords, numNewSamples * DIMENSION * sizeof(float)), "alloc d_coordsnew");
    check_error(hipMalloc(&d_distances, numNewSamples * numSamples * sizeof(float)), "alloc d_distances");

    //***copy-arrays-on-device***
    check_error(hipMemcpy(d_coords, coords, totalSamples * DIMENSION * sizeof(float), hipMemcpyHostToDevice), "copy d_coords");
    check_error(hipMemcpy(d_classes, classes, totalSamples * sizeof(int), hipMemcpyHostToDevice), "copy d_classes");
    check_error(hipMemcpy(d_newCoords, newCoords, numNewSamples * DIMENSION * sizeof(float), hipMemcpyHostToDevice), "copy d_coordsnew");

    const int PROBLEM_SIZE = numNewSamples;
    const int NUM_THREADS = 256;
    const int NUM_BLOCKS = (int)ceil(PROBLEM_SIZE / NUM_THREADS);

    // Calculate distances
    cuda_compute_distance<<<NUM_BLOCKS, NUM_THREADS>>>(d_coords, numSamples, d_newCoords, numNewSamples, d_distances);

    hipDeviceSynchronize();
    float *distances = new float[numNewSamples * numSamples];
    check_error(hipMemcpy(distances, d_distances, numNewSamples * numSamples * sizeof(float), hipMemcpyDeviceToHost), "download distances");
    hipDeviceSynchronize();
    for (int i = 0; i < numSamples * numNewSamples; i++)
    {
        printf("%f\t", distances[i]);
    }
    hipDeviceSynchronize();

    modified_insertion_sort<<<NUM_BLOCKS, NUM_THREADS>>>(
        d_distances,
        d_classes,
        numSamples,
        numNewSamples,
        k,
        numClasses
    );

    hipDeviceSynchronize();
    // download device -> host
    check_error(hipMemcpy(coords, d_coords, DIMENSION * totalSamples * sizeof(float), hipMemcpyDeviceToHost), "download coords");
    check_error(hipMemcpy(classes, d_classes, totalSamples * sizeof(int), hipMemcpyDeviceToHost), "download classes");
}

void check_error(hipError_t err, const char *msg)
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "%s : error %d (%s)\n", msg, err, hipGetErrorString(err));
        exit(err);
    }
}
