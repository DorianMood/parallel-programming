#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime_api.h>

#include "knn.cu"

void check_error(hipError_t err, const char *msg);
void knnParallel(float *coords, float *newCoords, int *classes, int numClasses, int numSamples, int numNewSamples, int k);

void knnParallel(float *coords, float *newCoords, int *classes, int numClasses, int numSamples, int numNewSamples, int k)
{
    //*** Device-variables-declaration ***
    float *d_coords;
    float *d_newCoords;
    int *d_classes;

    int totalSamples = numSamples + numNewSamples;

    //*** device-allocation ***
    check_error(hipMalloc(&d_coords, totalSamples * DIMENSION * sizeof(float)), "alloc d_coords_x");
    check_error(hipMalloc(&d_classes, totalSamples * sizeof(int)), "alloc d_classes");
    check_error(hipMalloc(&d_newCoords, numNewSamples * DIMENSION * sizeof(float)), "alloc d_coordsnew");

    //***copy-arrays-on-device***
    check_error(hipMemcpy(d_coords, coords, totalSamples * DIMENSION * sizeof(float), hipMemcpyHostToDevice), "copy d_coords");
    check_error(hipMemcpy(d_classes, classes, totalSamples * sizeof(int), hipMemcpyHostToDevice), "copy d_classes");
    check_error(hipMemcpy(d_newCoords, newCoords, numNewSamples * DIMENSION * sizeof(float), hipMemcpyHostToDevice), "copy d_coordsnew");

    const int PROBLEM_SIZE = numNewSamples;
    const int NUM_THREADS = 256;
    const int NUM_BLOCKS = (int)ceil(PROBLEM_SIZE / NUM_THREADS);

    knn<<<NUM_BLOCKS, NUM_THREADS>>>(
        d_coords,
        numSamples,
        d_newCoords,
        numNewSamples,
        d_classes,
        numClasses,
        k
    );

    hipDeviceSynchronize();
    // download device -> host
    check_error(hipMemcpy(coords, d_coords, DIMENSION * totalSamples * sizeof(float), hipMemcpyDeviceToHost), "download coords");
    check_error(hipMemcpy(classes, d_classes, totalSamples * sizeof(int), hipMemcpyDeviceToHost), "download classes");
}

void check_error(hipError_t err, const char *msg)
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "%s : error %d (%s)\n", msg, err, hipGetErrorString(err));
        exit(err);
    }
}
