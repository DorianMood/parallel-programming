#include "hip/hip_runtime.h"
#include <cstdio>

#include <>
#include <hip/hip_runtime.h>

#include <map>
#include <vector>
#include <math.h>

#include "base.h"

#define BLOCK_DIM 256

// extern "C"
// {
    /**
     * Compute distances from each B point to each A point
     * @param ref pointer to 1D feature refence vector
     * @param size_r size of vector reference
     * @param query pointer to 1D feature query vector of dimensions
     * @param size_q size of vector query
     * @param distance output vector of size_r * size_q
     * */
    __global__ void cuda_compute_distance(float *coords, int size_r,
                                          float *newCoords, int size_q,
                                          float *distance)
    {
        // Gloabal thread ID
        int tid = blockIdx.x * blockDim.x + threadIdx.x;

        int query_id = tid;

        if (tid < size_q)
        {
            // Get distances
            for (int i = 0; i < size_r; i++)
            {
                float sum = 0.0;
                for (int j = 0; j < DIMENSION; j++)
                {
                    sum += (coords[DIMENSION * i + j] - newCoords[tid * DIMENSION + j]) * (coords[DIMENSION * i + j] - newCoords[tid * DIMENSION + j]);
                }
                distance[tid * size_r + i] = sqrt(sum);
            }
        }
    }

    /**
    * For each reference point (i.e. each column) finds the k-th smallest distances
    * of the distance matrix and their respective indexes and gathers them at the top
    * of the 2 arrays.
    *
    * Since we only need to locate the k smallest distances, sorting the entire array
    * would not be very efficient if k is relatively small. Instead, we perform a
    * simple insertion sort by eventually inserting a given distance in the first
    * k values.
    *
    * @param dist         distance matrix
    * @param dist_pitch   pitch of the distance matrix given in number of columns
    * @param index        index matrix
    * @param index_pitch  pitch of the index matrix given in number of columns
    * @param width        width of the distance matrix and of the index matrix
    * @param height       height of the distance matrix
    * @param k            number of values to find
    */
    __global__ void modified_insertion_sort(float * dist,
                                            int *   classes,
                                            int     size_r,
                                            int     size_q,
                                            int     k,
                                            int     num_classes
    )
    {
        // Column position
        unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

        // For this query point
        if (xIndex < size_q)
        {
            // Pointer shift
            float * p_dist  = dist  + xIndex * size_r;

            // Initialise the top classes array
            int *top_classes = new int[k];
            for (int i = 0; i < k; i++)
                top_classes[i] = classes[size_r + i];

            // Iterate through all points
            for (int i = 1; i < size_r; ++i)
            {
                // Store current distance and associated index
                float curr_dist = p_dist[i];
                int curr_class = classes[i];

                // Skip the current value if its index is >= k and if it's higher the k-th already sorted mallest value
                if (curr_dist >= p_dist[(k - 1)])
                    continue;

                // Shift values (and indexes) higher that the current distance to the right
                int j = min(i, k - 1);
                while (j > 0 && p_dist[j - 1] > curr_dist)
                {
                    p_dist[j] = p_dist[j - 1];
                    top_classes[j] = top_classes[j - 1];
                    --j;
                }

                // Write the current distance and index at their position
                p_dist[j]   = curr_dist;
                top_classes[j] = curr_class; 
            }

            // Get class for current point accourding to the top classes
            // Implement majority vote
            classes[size_r + xIndex] = top_classes[0];
            int classes_sum = 0, classes_max = 0;
            for (int i = 0; i < num_classes; i++)
            {
                classes_sum = 0;
                for (int j = 0; j < k; j++)
                    if (i == top_classes[j])
                        classes_sum += 1;
                if (classes_sum > classes_max)
                {
                    classes[size_r + xIndex] = i;
                    classes_max = classes_sum;
                }
            }
        }
    }
//}
