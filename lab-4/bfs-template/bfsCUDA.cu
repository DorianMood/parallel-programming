
#include <hip/hip_runtime.h>

#include <cstdio>

extern "C" {
    // PUT YOUR KERNEL FUNCTION HERE
    __global__ void bfs_visit_next(
        int* adjacencyList,
        int* edgesOffset,
        int* edgesSize,
        int* distance,
        int* parent,
        int* currentQueue,
        int* nextQueue,
        int nextQueueSize,
        int* degrees,
        int* incrDegrees
    )
    {
        int tid = blockIdx.x * blockDim.x + threadIdx.x;

        int v = currentQueue[tid];
        for (int i = edgesOffset[v]; i < edgesOffset[v] + edgesSize[v]; i++)
        {
            int u = adjacencyList[i];
            
            if (parent[u] == -1) // Not visited
            { // Visit
                parent[u] = v;
                distance[u] = distance[v] + 1;

                nextQueue[nextQueueSize] = u;
                nextQueueSize++;
            }
        }
    __syncthreads();
    }
}
